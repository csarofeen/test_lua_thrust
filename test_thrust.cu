#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/random.h>

#include <iostream>

#include <hip/hip_fp16.h>

template <typename T>
struct asum_amax_type
{
  T asum_val;
  T amax_val;
  int nnz;
};

template <typename T>
struct asum_amax_binary_op
  : public thrust::binary_function< asum_amax_type<T>, asum_amax_type<T>, asum_amax_type<T> >
{
    __host__ __device__
    asum_amax_type<T> operator()(const asum_amax_type<T>& x, const asum_amax_type<T>& y) const
  {
    asum_amax_type<T> result;
    result.nnz = x.nnz + y.nnz;
    result.asum_val = x.asum_val + y.asum_val;
    result.amax_val = thrust::max(x.amax_val, y.amax_val);
    return result;
  }
};

struct h2f_unary_op
  : public thrust::unary_function<unsigned short, float>
{
  __device__
  asum_amax_type<float> operator()(const unsigned short& x) const
  {
    half val = *( (half*) &x);
    asum_amax_type<float> result;
    result.asum_val = fabsf(__half2float(val));
    result.amax_val = result.asum_val;
    result.nnz = (result.asum_val == 0.f) ? 0 : 1;
    return result;
  }
};

typedef struct float_pair{
  float aave;
  float amax;
} float_pair_t;

extern "C"
float_pair_t fp16_stats(half* d_data, int N){
  if((uintptr_t)(const void *)(d_data) % 4 == 0) std::cout<<"Aligned at 4Byte boundary"<<std::endl;
  else if( (uintptr_t)(const void *)(d_data) % 2 == 0) std::cout<<"Aligned at 2Byte boundary"<<std::endl;
  if(N%2 != 0){
    std::cerr<<"Odd sized tensors are not supported at the moment"<<std::endl;
    throw(-1);
  }

  thrust::device_ptr<unsigned short> d_ptr = thrust::device_pointer_cast((unsigned short*)d_data);

  h2f_unary_op unary_op;
  asum_amax_binary_op<float> binary_op;

  asum_amax_type<float> init;
  init.amax_val = 0;
  init.nnz=0;
  init.asum_val = 0;

  asum_amax_type<float> result = thrust::transform_reduce(d_ptr, d_ptr+N, unary_op, init, binary_op);
  float_pair_t return_result;
  return_result.aave = result.asum_val/(float)result.nnz;
  return_result.amax = result.amax_val;
  std::cout<<return_result.aave<<std::endl;
  std::cout<<return_result.amax<<std::endl;
  return return_result;

}


